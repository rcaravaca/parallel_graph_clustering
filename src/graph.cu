#include "hip/hip_runtime.h"
#include "graph.h"

Graph::Graph(){};

// void Graph::addEdge(const Digit& source, const Digit& destination) {
//     // Buscar el nodo fuente en adjList
//     for (auto& nodeList : adjList) {
//         if (nodeList.front().getID() == source.getID()) {
//             nodeList.push_back(destination);  // Add the destination node to neighbors list
//             return;
//         }
//     }

//     std::cerr << "Error: Source node not founded." << std::endl;
// }

void Graph::addEdge(const Digit& source, const Digit& destination, int weight) {
    // Search for the source node in adjList
    for (auto& nodeList : adjList) {
        if (nodeList.front().first.getID() == source.getID()) {
            nodeList.push_back({destination, weight});  // Add the destination node with the weight
            return;
        }
    }

    // If the source node is not found, print an error
    std::cerr << "Error: Source node not found." << std::endl;
}

// void Graph::addNode(const Digit& digit) {
//     adjList.push_back(std::vector<Digit>{digit});  // the first elemnet is the node, without neighbors
// }

void Graph::addNode(const Digit& newNode) {
    // Check if the node already exists in the graph (optional)
    for (const auto& nodeList : adjList) {
        if (nodeList.front().first.getID() == newNode.getID()) {
            std::cerr << "Error: Node with ID " << newNode.getID() << " already exists in the graph." << std::endl;
            return;
        }
    }

    // Add the new node to the graph with no neighbors (self-weight 0)
    adjList.push_back({{newNode, 0}});
}

bool Graph::nodeExists(const Digit& digit) const {
    for (const auto& nodeList : adjList) {
        if (nodeList.front().first.getID() == digit.getID()) {
            return true;
        }
    }
    return false;  // Node not founded
}

const Digit& Graph::getNode(int index) const {
    if (index >= 0 && index < adjList.size()) {
        return adjList[index].front().first;  // The first element in internal vector is the node
    } else {
        throw std::out_of_range("Index out of range.");
    }
}

// void Graph::printGraph() const {
//     for (const auto& nodeList : adjList) {
//         const Digit& node = nodeList.front();
//         std::cout << "Node " << node.getID() << " (Row=" << node.getRow() 
//                   << ", Col=" << node.getCol() << ", Energy=" << node.getEnergy() << "): ";

//         for (size_t i = 1; i < nodeList.size(); ++i) {
//             const Digit& neighbor = nodeList[i];
//             std::cout << "Digit(ID=" << neighbor.getID() << ", Row=" << neighbor.getRow()
//                       << ", Col=" << neighbor.getCol() << ", Energy=" << neighbor.getEnergy() << ") ";
//         }
//         std::cout << std::endl;
//     }
// }

void Graph::printGraph() const {
    int count = 0;  // Counter to keep track of the number of nodes printed

    // Loop through each node's adjacency list
    for (const auto& nodeList : adjList) {
        if (count >= 10) {
            break;  // Stop after printing 10 nodes
        }

        // The first element in the nodeList is the current node
        const Digit& node = nodeList.front().first;

        // Print the current node
        std::cout << "Node " << node.getID() << " (Row: " << node.getRow() 
                  << ", Col: " << node.getCol() << ", Energy: " << node.getEnergy() << ") -> ";

        // Print all its neighbors
        for (size_t i = 1; i < nodeList.size(); ++i) {
            const Digit& neighbor = nodeList[i].first;
            int weight = nodeList[i].second;  // Retrieve the weight for this neighbor

            // Print the neighbor and the edge weight
            std::cout << "Neighbor " << neighbor.getID() << " (Weight: " << weight << "), ";
        }

        std::cout << std::endl;  // End of this node's adjacency list

        count++;  // Increment the counter after printing a node
    }

    if (count == 0) {
        std::cout << "Graph is empty or has no nodes to print." << std::endl;
    }
}

void Graph::flattenGraph(std::vector<int>& flatAdjList, std::vector<int>& adjListSizes, std::vector<int>& Nodes, int& numNodes) const {

    numNodes = adjList.size();

    for (const auto& nodeList : adjList) {
        
        // nodes.push_back(nodeList.front().getID());
        const Digit& node = nodeList.front().first;  // First element is the node itself
        
        Nodes.push_back(node.getRow());
        Nodes.push_back(node.getCol());
        Nodes.push_back(node.getEnergy());

        Nodes.push_back(nodeList.front().second);
        
        adjListSizes.push_back(flatAdjList.size()); 
        for (size_t i = 1; i < nodeList.size(); ++i) {
            // flatAdjList.push_back(nodeList[i].getID());
            // flatAdjList.push_back(nodeList[i].getRow());
            // flatAdjList.push_back(nodeList[i].getCol());
            // flatAdjList.push_back(nodeList[i].getEnergy());

            const Digit& neighbor = nodeList[i].first;  // Neighbor and weight
            flatAdjList.push_back(neighbor.getRow());
            flatAdjList.push_back(neighbor.getCol());
            flatAdjList.push_back(neighbor.getEnergy());
            flatAdjList.push_back(nodeList[i].second);  // Include the weight in the same array
        }
    }
}

void Graph::rebuildGraph(const std::vector<int>& flatAdjList, const std::vector<int>& adjListSizes, const std::vector<int>& Nodes, int numNodes) {
    
    adjList.clear();  // clean the graph

    int adjIndex = 0;  // Index for flatAdjList

    // Rebuild the graph
    for (int i = 0; i < numNodes; ++i) {
        
        // Every node has 3 values: row, col & energy
        int row = Nodes[i * 4]; 
        int col = Nodes[i * 4 + 1];
        int energy = Nodes[i * 4 + 2];

        // std::cout << "Nodo 0: Row=" << row << ", Col=" << col << ", Energy=" << energy << std::endl;

        // Make then main node and add to graph
        Digit node(row, col, energy);
        adjList.push_back({{node, 0}});  // The first entry in the list, weight is 0 for the node itself
        // addNode(node);

        // Get the neighbor of node
        int startIdx = adjListSizes[i];  // Begining index of flatAdjList
        int endIdx = (i + 1 < adjListSizes.size()) ? adjListSizes[i + 1] : flatAdjList.size() / 4;  // Adjust for 4 entries per neighbor

        // add the neighbors
        if (startIdx < endIdx) {
            for (int j = startIdx; j < endIdx; ++j) {

                int neighborRow = flatAdjList[adjIndex++];
                int neighborCol = flatAdjList[adjIndex++];
                int neighborEnergy = flatAdjList[adjIndex++];
                int weight = flatAdjList[adjIndex++];  // Retrieve the weight
                
                Digit neighbor(neighborRow, neighborCol, neighborEnergy);
                adjList.back().push_back({neighbor, weight});  // Add neighbor and weight
                // addEdge(node, neighbor, weight);

            }
        }
    }
}

void Graph::GraphSummary() const {
    // Verificar si el grafo está vacío
    if (adjList.empty()) {
        std::cout << "Graph is empty." << std::endl;
        return;
    }

    // Variables para los cálculos
    int numNodes = 0;
    int minEnergy = INT_MAX;
    int maxEnergy = INT_MIN;
    double sumEnergy = 0;
    double sumEnergySquared = 0;  // Para calcular la desviación estándar

    // Recorrer todos los nodos y calcular los valores de energía
    for (const auto& nodeList : adjList) {
        if (!nodeList.empty()) {
            const Digit& node = nodeList.front().first;  // El nodo principal de la lista de adyacencia

            int energy = node.getEnergy();
            numNodes++;

            // Actualizar el valor mínimo y máximo de energía
            if (energy < minEnergy) {
                minEnergy = energy;
            }
            if (energy > maxEnergy) {
                maxEnergy = energy;
            }

            // Acumular la suma de las energías y la suma de las energías al cuadrado
            sumEnergy += energy;
            sumEnergySquared += energy * energy;
        }
    }

    // Cálculos finales
    double meanEnergy = sumEnergy / numNodes;
    double variance = (sumEnergySquared / numNodes) - (meanEnergy * meanEnergy);
    double stdDeviation = sqrt(variance);

    // Imprimir el resumen del grafo
    std::cout << "\n##############################" << std::endl;
    std::cout << "Graph Summary:" << std::endl;
    std::cout << "Number of nodes: " << numNodes << std::endl;
    std::cout << "Energy values: " << std::endl;
    std::cout << "  Minimum: " << minEnergy << std::endl;
    std::cout << "  Maximum: " << maxEnergy << std::endl;
    std::cout << "  Average: " << meanEnergy << std::endl;
    std::cout << "  Standard deviation: " << stdDeviation << "\n##############################" << std::endl<< std::endl;
}


bool Graph::checkForDuplicateIDs() const {

    std::unordered_set<int> seenIDs;  // To track the IDs we have already encountered

    // Traverse the adjacency list
    for (const auto& nodeList : adjList) {
        // The first element in the nodeList is the current node
        const Digit& node = nodeList.front().first;

        // Check if this node's ID has already been seen
        if (seenIDs.find(node.getID()) != seenIDs.end()) {
            // Duplicate ID found, print the duplicate and return true
            std::cerr << "Duplicate ID found: Node " << node.getID() << std::endl;
            return true;
        }

        // If not seen, add this node's ID to the set
        seenIDs.insert(node.getID());
    }

    // If we reach here, no duplicate IDs were found
    std::cout << "No duplicate node IDs found." << std::endl;
    return false;
}
