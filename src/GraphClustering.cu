#include "GraphClustering.h"
#include "kernels/processGraph.h"

void GraphInsertion(Graph& graph, int maxNodes, const std::vector<Digit>& digits) {

    std::cout << "GraphInsertion: Starting the Graph Nodes Insertion..." << std::endl;

    std::vector<int> flatAdjList;
    std::vector<int> adjListSizes;
    std::vector<int> Nodes;
    int numNodes;

    // Aplanar el grafo en arrays
    graph.flattenGraph(flatAdjList, adjListSizes, Nodes, numNodes);

    // Crear arrays planos para los datos de los Digits
    int numDigits = digits.size();
    std::vector<int> rows(numDigits);
    std::vector<int> cols(numDigits);
    std::vector<int> energies(numDigits);

    for (int i = 0; i < numDigits; ++i) {
        rows[i] = digits[i].getRow();
        cols[i] = digits[i].getCol();
        energies[i] = digits[i].getEnergy();
    }

    // Calcular el tamaño necesario para Nodes y adjList
    int Nodesize = maxNodes * 3;       // 3 enteros por nodo
    int adjListSize = maxNodes * 9 * 3;  // Hasta 9 vecinos por nodo, cada uno con 3 enteros

    Nodes.resize(Nodesize);
    adjListSizes.resize(Nodesize);
    flatAdjList.resize(adjListSize);

    // Asignar memoria en el dispositivo
    int* d_adjList, *d_adjListSizes, *d_Nodes, *d_numNodes, *d_rows, *d_cols, *d_energies;
    hipMalloc(&d_adjList, adjListSize * sizeof(int));  // Reservar suficiente espacio para adjList
    hipMalloc(&d_adjListSizes, maxNodes * sizeof(int));  // Una entrada por cada nodo
    hipMalloc(&d_Nodes, Nodesize * sizeof(int));  // Reservar suficiente espacio para los nodos
    hipMalloc(&d_numNodes, sizeof(int));  // Un entero para el número de nodos
    hipMalloc(&d_rows, rows.size() * sizeof(int));  // Filas de los Digits
    hipMalloc(&d_cols, cols.size() * sizeof(int));  // Columnas de los Digits
    hipMalloc(&d_energies, energies.size() * sizeof(int));  // Energías de los Digits

    // Copiar datos al dispositivo
    hipMemcpy(d_adjList, flatAdjList.data(), adjListSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), maxNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Nodes, Nodes.data(), Nodesize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);  // Inicializar numNodes a 0 en el dispositivo
    hipMemcpy(d_rows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, cols.data(), cols.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_energies, energies.data(), energies.size() * sizeof(int), hipMemcpyHostToDevice);


    // Configuración del kernel
    int blockSize = 256;
    int numBlocks = (numDigits + blockSize - 1) / blockSize;  // Calcular número de bloques

    // Lanzar el kernel para agregar los nodos
    addNodeToGraphCUDA<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_Nodes, d_numNodes, maxNodes, d_rows, d_cols, d_energies, numDigits);

    // Sincronizar el dispositivo
    hipDeviceSynchronize();

    // Copiar resultados de vuelta al host
    hipMemcpy(flatAdjList.data(), d_adjList, adjListSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(adjListSizes.data(), d_adjListSizes, maxNodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Nodes.data(), d_Nodes, Nodesize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);  // Verificar que numNodes se actualice

    std::cout << "GraphInsertion: Count of added Nodes: " << numNodes << std::endl;

    // Reconstruir el grafo en el host
    graph.rebuildGraph(flatAdjList, adjListSizes, Nodes, numNodes);

    // Liberar memoria
    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_Nodes);
    hipFree(d_numNodes);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_energies);
    
    std::cout << "GraphInsertion: Done" << std::endl;

}
