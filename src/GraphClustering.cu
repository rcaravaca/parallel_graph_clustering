#include "GraphClustering.h"
#include "kernels/processGraph.h"

void GraphInsertion(Graph& graph, int maxNodes, const std::vector<Digit>& digits) {

    std::cout << "GraphInsertion: Starting the Graph Nodes Insertion..." << std::endl;

    std::vector<int> flatAdjList;
    std::vector<int> adjListSizes;
    std::vector<int> Nodes;
    std::vector<int> flatWeights;
    int numNodes;

    // Aplanar el grafo en arrays
    graph.flattenGraph(flatAdjList, adjListSizes, Nodes, numNodes);

    // Crear arrays planos para los datos de los Digits
    int numDigits = digits.size();
    std::vector<int> rows(numDigits);
    std::vector<int> cols(numDigits);
    std::vector<int> energies(numDigits);

    for (int i = 0; i < numDigits; ++i) {
        rows[i] = digits[i].getRow();
        cols[i] = digits[i].getCol();
        energies[i] = digits[i].getEnergy();
    }

    // Calcular el tamaño necesario para Nodes y adjList
    int NodeSize = maxNodes * 4;       // 3 enteros por nodo + 1 weight
    int adjListSize = maxNodes * 8 * 4;  // Hasta 8 vecinos por nodo, cada uno con 4 enteros
    int weightSize = maxNodes * 8;     // 1 weight per neighbor, up to 8 neighbors per node

    Nodes.resize(NodeSize);
    adjListSizes.resize(NodeSize);
    flatAdjList.resize(adjListSize);
    flatWeights.resize(weightSize);

    // Asignar memoria en el dispositivo
    int *d_adjList, *d_adjListSizes, *d_Nodes, *d_numNodes, *d_rows, *d_cols, *d_energies, *d_flatWeights;
    hipMalloc(&d_adjList, adjListSize * sizeof(int));  // Reservar suficiente espacio para adjList
    hipMalloc(&d_adjListSizes, maxNodes * sizeof(int));  // Una entrada por cada nodo
    hipMalloc(&d_Nodes, NodeSize * sizeof(int));  // Reservar suficiente espacio para los nodos
    hipMalloc(&d_numNodes, sizeof(int));  // Un entero para el número de nodos
    hipMalloc(&d_rows, rows.size() * sizeof(int));  // Filas de los Digits
    hipMalloc(&d_cols, cols.size() * sizeof(int));  // Columnas de los Digits
    hipMalloc(&d_energies, energies.size() * sizeof(int));  // Energías de los Digits
    hipMalloc(&d_flatWeights, weightSize * sizeof(int));  // Weights of the edges

    // Copiar datos al dispositivo
    hipMemcpy(d_adjList, flatAdjList.data(), adjListSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), maxNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Nodes, Nodes.data(), NodeSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);  // Inicializar numNodes a 0 en el dispositivo
    hipMemcpy(d_rows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, cols.data(), cols.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_energies, energies.data(), energies.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flatWeights, flatWeights.data(), flatWeights.size() * sizeof(int), hipMemcpyHostToDevice);


    // Configuración del kernel
    int blockSize = 256;
    int numBlocks = (numDigits + blockSize - 1) / blockSize;  // Calcular número de bloques

    // Lanzar el kernel para agregar los nodos
    addNodeToGraphCUDA<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_Nodes, d_numNodes, maxNodes, d_rows, d_cols, d_energies, numDigits, d_flatWeights);
    // Sincronizar el dispositivo
    hipDeviceSynchronize();

    // Copiar resultados de vuelta al host
    hipMemcpy(flatAdjList.data(), d_adjList, adjListSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(adjListSizes.data(), d_adjListSizes, maxNodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Nodes.data(), d_Nodes, NodeSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(flatWeights.data(), d_flatWeights, weightSize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "GraphInsertion: Count of added Nodes: " << numNodes << std::endl;

    // Reconstruir el grafo en el host
    graph.rebuildGraph(flatAdjList, adjListSizes, Nodes, numNodes);

    // Liberar memoria
    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_Nodes);
    hipFree(d_numNodes);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_energies);
    hipFree(d_flatWeights);
    
    std::cout << "GraphInsertion: Done" << std::endl;

}
