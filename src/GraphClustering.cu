#include "GraphClustering.h"
#include "kernels/processGraph.h"

void GraphInsertion(Graph& graph, int maxNodes, const std::vector<Digit>& digits) {

    std::cout << "GraphInsertion: Starting the Graph Nodes Insertion..." << std::endl;

    std::vector<int> flatAdjList;
    std::vector<int> adjListSizes;
    std::vector<int> Nodes;
    int numNodes;

    // Flatten the graph into arrays
    graph.flattenGraph(flatAdjList, adjListSizes, Nodes, numNodes); // TODO: What is this for? This is not doing anything atm

    // Create flat arrays for the Digit data
    int numDigits = digits.size();
    std::vector<int> rows(numDigits);
    std::vector<int> cols(numDigits);
    std::vector<int> energies(numDigits);

    for (int i = 0; i < numDigits; ++i) {
        rows[i] = digits[i].getRow();
        cols[i] = digits[i].getCol();
        energies[i] = digits[i].getEnergy();
    }

    // Calculate the necessary size for Nodes and adjList
    int NodeSize = maxNodes * 3;       // 3 integers per node (row, col, energy)
    int adjListSize = maxNodes * 8 * 4;  // Up to 8 neighbors per node, each with 4 integers (row, col, energy, weight of the edge)

    Nodes.resize(NodeSize);
    adjListSizes.resize(NodeSize);
    flatAdjList.resize(adjListSize);

    // Allocate memory on the device
    int *d_adjList, *d_adjListSizes, *d_Nodes, *d_numNodes, *d_rows, *d_cols, *d_energies;
    hipMalloc(&d_adjList, adjListSize * sizeof(int));  // Allocate enough space for adjList
    hipMalloc(&d_adjListSizes, maxNodes * sizeof(int));  // One entry per node
    hipMalloc(&d_Nodes, NodeSize * sizeof(int));  // Allocate enough space for nodes
    hipMalloc(&d_numNodes, sizeof(int));  // One integer for the number of nodes
    hipMalloc(&d_rows, rows.size() * sizeof(int));  // Rows of the Digits
    hipMalloc(&d_cols, cols.size() * sizeof(int));  // Columns of the Digits
    hipMalloc(&d_energies, energies.size() * sizeof(int));  // Energies of the Digits

    // Copy data to the device
    hipMemcpy(d_adjList, flatAdjList.data(), adjListSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), maxNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Nodes, Nodes.data(), NodeSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);  // Initialize numNodes to 0 on the device
    hipMemcpy(d_rows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, cols.data(), cols.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_energies, energies.data(), energies.size() * sizeof(int), hipMemcpyHostToDevice);

    // Kernel configuration
    int blockSize = 256;
    int numBlocks = (numDigits + blockSize - 1) / blockSize;  // Calculate number of blocks

    // Launch the kernel to add nodes
    addNodeToGraphCUDA<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_Nodes, d_numNodes, maxNodes, d_rows, d_cols, d_energies, numDigits);
    // Synchronize the device
    hipDeviceSynchronize();

    // Copy results back to the host
    hipMemcpy(flatAdjList.data(), d_adjList, adjListSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(adjListSizes.data(), d_adjListSizes, maxNodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Nodes.data(), d_Nodes, NodeSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "GraphInsertion: Count of added Nodes: " << numNodes << std::endl;

    // Rebuild the graph on the host
    graph.rebuildGraph(flatAdjList, adjListSizes, Nodes, numNodes);

    // Free memory
    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_Nodes);
    hipFree(d_numNodes);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_energies);
    
    std::cout << "GraphInsertion: Done" << std::endl;

}
