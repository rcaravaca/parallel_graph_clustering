#include "GraphClustering.h"
#include "kernels/processGraph.h"

void GraphInsertion(Graph& graph, int maxNodes, const std::vector<Digit>& digits) {

    std::cout << "GraphInsertion: Starting the Graph Nodes Insertion..." << std::endl;

    std::vector<int> flatAdjList;
    std::vector<int> adjListSizes;
    std::vector<int> Nodes;
    std::vector<int> flatWeights;
    int numNodes;

    // Flatten the graph into arrays
    graph.flattenGraph(flatAdjList, adjListSizes, Nodes, numNodes); // TODO: What is this for? This is not doing anything atm

    // Create flat arrays for the Digit data
    int numDigits = digits.size();
    std::vector<int> rows(numDigits);
    std::vector<int> cols(numDigits);
    std::vector<int> energies(numDigits);

    for (int i = 0; i < numDigits; ++i) {
        rows[i] = digits[i].getRow();
        cols[i] = digits[i].getCol();
        energies[i] = digits[i].getEnergy();
    }

    // Calculate the necessary size for Nodes and adjList
    int NodeSize = maxNodes * 3;       // 3 integers per node (row, col, energy)
    int adjListSize = maxNodes * 8 * 3;  // Up to 8 neighbors per node, each with 3 integers (row, col, energy)
    int weightSize = maxNodes * 8; // Up to 8 neighbors per node, each with 1 value (weight of the edge)

    Nodes.resize(NodeSize);
    adjListSizes.resize(maxNodes);
    flatAdjList.resize(adjListSize);
    flatWeights.resize(weightSize);

    // Allocate memory on the device
    int *d_adjList, *d_adjListSizes, *d_Nodes, *d_numNodes, *d_rows, *d_cols, *d_energies, *d_flatWeights;
    hipMalloc(&d_adjList, adjListSize * sizeof(int));  // Allocate enough space for adjList
    hipMalloc(&d_adjListSizes, maxNodes * sizeof(int));  // One entry per node
    hipMalloc(&d_Nodes, NodeSize * sizeof(int));  // Allocate enough space for nodes
    hipMalloc(&d_numNodes, sizeof(int));  // One integer for the number of nodes
    hipMalloc(&d_rows, rows.size() * sizeof(int));  // Rows of the Digits
    hipMalloc(&d_cols, cols.size() * sizeof(int));  // Columns of the Digits
    hipMalloc(&d_energies, energies.size() * sizeof(int));  // Energies of the Digits
    hipMalloc(&d_flatWeights, weightSize * sizeof(int));  // Weights of the edges

    // Copy data to the device
    hipMemcpy(d_adjList, flatAdjList.data(), adjListSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), maxNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Nodes, Nodes.data(), NodeSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numNodes, &numNodes, sizeof(int), hipMemcpyHostToDevice);  // Initialize numNodes to 0 on the device
    hipMemcpy(d_rows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, cols.data(), cols.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_energies, energies.data(), energies.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flatWeights, flatWeights.data(), flatWeights.size() * sizeof(int), hipMemcpyHostToDevice);

    // Kernel configuration
    int blockSize = 256;
    int numBlocks = (numDigits + blockSize - 1) / blockSize;  // Calculate number of blocks

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the kernel to add nodes
    hipEventRecord(start);
    addNodeToGraphCUDA<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_Nodes, d_numNodes, maxNodes, d_rows, d_cols, d_energies, numDigits, d_flatWeights);
    hipEventRecord(stop);
    
    // Synchronize the device
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time: " << milliseconds << " ms" << std::endl;

    // Copy results back to the host
    hipMemcpy(flatAdjList.data(), d_adjList, adjListSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(adjListSizes.data(), d_adjListSizes, maxNodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Nodes.data(), d_Nodes, NodeSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numNodes, d_numNodes, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(flatWeights.data(), d_flatWeights, weightSize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "GraphInsertion: Count of added Nodes: " << numNodes << std::endl;

    // Rebuild the graph on the host
    graph.rebuildGraph(flatAdjList, adjListSizes, flatWeights, Nodes, numNodes);

    // Free memory
    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_Nodes);
    hipFree(d_numNodes);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_energies);
    hipFree(d_flatWeights);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    std::cout << "GraphInsertion: Done" << std::endl;

}

void GraphInsertionV2(Graph& graph, int maxNodes, const std::vector<Digit>& digits) {

    std::cout << "GraphInsertionV2: Starting the Graph Nodes Insertion..." << std::endl;

    std::vector<int> flatAdjList;
    std::vector<int> adjListSizes;
    std::vector<int> Seeds;
    std::vector<int> flatWeights;
    int numSeeds;

    // Create flat arrays for the Digit data
    int numDigits = digits.size();
    std::vector<int> rows(numDigits);
    std::vector<int> cols(numDigits);
    std::vector<int> energies(numDigits);

    for (int i = 0; i < numDigits; ++i) {
        rows[i] = digits[i].getRow();
        cols[i] = digits[i].getCol();
        energies[i] = digits[i].getEnergy();
    }

    // Calculate the necessary size for Nodes and adjList
    int SeedSize = maxNodes * 3;       // 3 integers per node (row, col, energy)
    int adjListSize = maxNodes * 8 * 3;  // Up to 8 neighbors per node, each with 3 integers (row, col, energy)
    int weightSize = maxNodes * 8; // Up to 8 neighbors per node, each with 1 value (weight of the edge)

    Seeds.resize(SeedSize);
    adjListSizes.resize(maxNodes);
    flatAdjList.resize(adjListSize);
    flatWeights.resize(weightSize);

    // Allocate memory on the device
    int *d_adjList, *d_adjListSizes, *d_Seeds, *d_numSeeds, *d_rows, *d_cols, *d_energies, *d_flatWeights;
    hipMalloc(&d_adjList, adjListSize * sizeof(int));  // Allocate enough space for adjList
    hipMalloc(&d_adjListSizes, maxNodes * sizeof(int));  // One entry per node
    hipMalloc(&d_Seeds, SeedSize * sizeof(int));  // Allocate enough space for nodes
    hipMalloc(&d_numSeeds, sizeof(int));  // One integer for the number of nodes
    hipMalloc(&d_rows, rows.size() * sizeof(int));  // Rows of the Digits
    hipMalloc(&d_cols, cols.size() * sizeof(int));  // Columns of the Digits
    hipMalloc(&d_energies, energies.size() * sizeof(int));  // Energies of the Digits
    hipMalloc(&d_flatWeights, weightSize * sizeof(int));  // Weights of the edges

    numSeeds = 0;  // Initialize the number of seeds to 0

    // Copy data to the device
    hipMemcpy(d_adjList, flatAdjList.data(), adjListSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjListSizes, adjListSizes.data(), maxNodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Seeds, Seeds.data(), SeedSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numSeeds, &numSeeds, sizeof(int), hipMemcpyHostToDevice);  // Initialize numSeeds to 0 on the device
    hipMemcpy(d_rows, rows.data(), rows.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cols, cols.data(), cols.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_energies, energies.data(), energies.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_flatWeights, flatWeights.data(), flatWeights.size() * sizeof(int), hipMemcpyHostToDevice);

    // Kernel configuration
    dim3 blockSize = dim3(8, 32); // will use 8 threads for each digit
    int numBlocks = (numDigits + 32 - 1) / 32;  // Calculate number of blocks

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    // Launch the kernel to add nodes
    addNodeToGraphCUDAv2<<<numBlocks, blockSize>>>(d_adjList, d_adjListSizes, d_Seeds, d_numSeeds, maxNodes, d_rows, d_cols, d_energies, numDigits, d_flatWeights);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time: " << milliseconds << " ms" << std::endl;

    // Synchronize the device

    // Copy results back to the host
    hipMemcpy(flatAdjList.data(), d_adjList, adjListSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(adjListSizes.data(), d_adjListSizes, maxNodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(Seeds.data(), d_Seeds, SeedSize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&numSeeds, d_numSeeds, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(flatWeights.data(), d_flatWeights, weightSize * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "GraphInsertion: Count of added Seeds: " << numSeeds << std::endl;

    // Rebuild the graph on the host
    graph.rebuildGraph(flatAdjList, adjListSizes, flatWeights, Seeds, numSeeds);

    // Free memory
    hipFree(d_adjList);
    hipFree(d_adjListSizes);
    hipFree(d_Seeds);
    hipFree(d_numSeeds);
    hipFree(d_rows);
    hipFree(d_cols);
    hipFree(d_energies);
    hipFree(d_flatWeights);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    std::cout << "GraphInsertion: Done" << std::endl;

}
